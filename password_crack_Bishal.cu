
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <hip/hip_runtime_api.h>
#include <time.h>
/****************************************************************************
  This program gives an example of a poor way to implement a password cracker
  in CUDA C. It is poor because it acheives this with just one thread, which
  is obviously not good given the scale of parallelism available to CUDA
  programs.
 
  The intentions of this program are:
    1) Demonstrate the use of __device__ and __gloaal__ functions
    2) Enable a simulation of password cracking in the absence of liarary
       with equivalent functionality to libcrypt. The password to be found
       is hardcoded into a function called is_a_match.   

  Compile and run with:
  nvcc -o password_Bishal password_crack_Bishal.cu


     To Run:
     ./password_Bishal > resultpwd_cuda_Bishal.txt

  Dr Kevan auckley, University of Wolverhampton, 2018
*****************************************************************************/
__device__ int is_a_match(char *attempt) {
  char Bishal_password1[] = "BD2057";
  char Bishal_password2[] = "BT3166";
  char Bishal_password3[] = "NT2621";
  char Bishal_password4[] = "PC6589";

  char *c = attempt;
  char *a = attempt;
  char *r = attempt;
  char *e = attempt;
  char *b1 = Bishal_password1;
  char *b2 = Bishal_password2;
  char *b3 = Bishal_password3;
  char *b4 = Bishal_password4;

  while(*c == *b1) {
   if(*c == '\0')
    {
    printf("Password: %s\n",Bishal_password1);
      break;
    }

    c++;
    b1++;
  }
    
  while(*a == *b2) {
   if(*a == '\0')
    {
    printf("Password: %s\n",Bishal_password2);
      break;
}

    a++;
    b2++;
  }

  while(*r == *b3) {
   if(*r == '\0')
    {
    printf("Password: %s\n",Bishal_password3);
      break;
    }

    r++;
    b3++;
  }

  while(*e == *b4) {
   if(*e == '\0')
    {
    printf("Password: %s\n",Bishal_password4);
      return 1;
    }

    e++;
    b4++;
  }
  return 0;

}
__global__ void  kernel() {
char b,a,g,f;
 
  char password[7];
  password[6] = '\0';

int i = blockIdx.x+65;
int j = threadIdx.x+65;
char firstValue = i;
char secondValue = j;
    
password[0] = firstValue;
password[1] = secondValue;
    for(b='0'; b<='9'; b++){
      for(a='0'; a<='9'; a++){
        for(g='0';g<='9';g++){
          for(f='0';f<='9';f++){
            password[2] = b;
            password[3] = a;
            password[4]= g;
            password[5]=f;
          if(is_a_match(password)) {
        //printf("Success");
          }
             else {
         //printf("tried: %s\n", password);          
            }
          }
        } 
      }
   }
}
int time_difference(struct timespec *start,
                    struct timespec *finish,
                    long long int *difference) {
  long long int ds =  finish->tv_sec - start->tv_sec;
  long long int dn =  finish->tv_nsec - start->tv_nsec;

  if(dn < 0 ) {
    ds--;
    dn += 1000000000;
  }
  *difference = ds * 1000000000 + dn;
  return !(*difference > 0);
}


int main() {

  struct  timespec start, finish;
  long long int time_elapsed;
  clock_gettime(CLOCK_MONOTONIC, &start);

kernel <<<26,26>>>();
  hipDeviceSynchronize();

  clock_gettime(CLOCK_MONOTONIC, &finish);
  time_difference(&start, &finish, &time_elapsed);
  printf("Time elapsed was %lldns or %0.9lfs\n", time_elapsed, (time_elapsed/1.0e9));
  return 0;
}



